#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
//                     The LLVM Compiler Infrastructure
//
// This file is dual licensed under the MIT and the University of Illinois Open
// Source Licenses. See LICENSE.TXT for details.
//
//===----------------------------------------------------------------------===//

#include <assert.h>
#include <stdio.h>
#include <complex>

// These are loosely adapted from libc++'s tests.  In general, we don't care a
// ton about verifying the return types or results we get, on the assumption
// that our standard library is correct. But we care deeply about calling every
// overload of every function (so that we verify that everything compiles).
//
// We do care about the results of complex multiplication / division, since
// these use code we've written.

// These tests are pretty annoying to write without C++11, so we require that.
// In addition, these tests currently don't compile with libc++, because of the
// issue in https://reviews.llvm.org/D25403.
//
// TODO: Once that issue is resolved, take out !defined(_LIBCPP_VERSION) here.
#if __cplusplus >= 201103L && !defined(_LIBCPP_VERSION)

#include <type_traits>

template <class T>
__device__ double promote(
    T, typename std::enable_if<std::is_integral<T>::value>::type* = 0);
__device__ float promote(float);
__device__ double promote(double);

__device__ void is_about(float x, float y) {
  assert(std::abs((x - y) / (x + y)) < 1.e-6);
}
__device__ void is_about(double x, double y) {
  assert(std::abs((x - y) / (x + y)) < 1.e-14);
}

template <class T>
__device__ void test_promotion_impl(T x) {
  assert(std::imag(x) == 0);
  assert(std::real(x) == x);

  using Promoted = decltype(promote(x));
  assert(std::arg(x) == arg(std::complex<Promoted>(x, 0)));
  assert(std::conj(x) == conj(std::complex<Promoted>(x, 0)));
  assert(std::norm(x) == norm(std::complex<Promoted>(x, 0)));
#ifndef __GLIBCXX__
  // libstdc++'s implementation of proj is completely broken, see
  // https://gcc.gnu.org/bugzilla/show_bug.cgi?id=61761.
  assert(std::proj(x) == proj(std::complex<Promoted>(x, 0)));
#endif
}

__device__ void test_promotion() {
  int vals[] = {0, 1, 10};
  for (int i : vals) {
    test_promotion_impl<float>(i);
    test_promotion_impl<double>(i);
    test_promotion_impl<int>(i);
    test_promotion_impl<unsigned>(i);
    test_promotion_impl<long long>(i);
  }
}

__device__ void test_literals() {
#if __cplusplus >= 201402L
  using namespace std::literals::complex_literals;

  {
    std::complex<double> c1 = 3.0i;
    assert(c1 == std::complex<double>(0, 3.0));
    auto c2 = 3i;
    assert(c1 == c2);
  }

  {
    std::complex<float> c1 = 3.0if;
    assert(c1 == std::complex<float>(0, 3.0));
    auto c2 = 3if;
    assert(c1 == c2);
  }
#endif
}

template <class T>
__device__ void test_assignment_real() {
  std::complex<T> c;
  c = 1.5;
  assert(c.real() == 1.5);
  assert(c.imag() == 0);
}

template <class T, class U>
__device__ void test_assignment_complex() {
  std::complex<T> c;
  std::complex<T> c2(1.5, 2.5);
  c = c2;
  assert(c.real() == 1.5);
  assert(c.imag() == 2.5);
}

template <class T>
__device__ void test_plus_equals() {
  {
    std::complex<T> c;
    c += 1.5;
    assert(c.real() == 1.5);
    assert(c.imag() == 0);
  }

  {
    std::complex<T> c;
    const std::complex<T> c2(1.5, 2.5);
    c += c2;
    c += c2;
    assert(c.real() == 3);
    assert(c.imag() == 5);

    std::complex<T> c3;

    c3 = c;
    std::complex<int> ic(1, 1);
    c3 += ic;
    assert(c3.real() == 4);
    assert(c3.imag() == 6);

    c3 = c;
    std::complex<float> fc(1, 1);
    c3 += fc;
    assert(c3.real() == 4);
    assert(c3.imag() == 6);
  }
}

template <class T>
__device__ void test_minus_equals() {
  {
    std::complex<T> c;
    c -= 1.5;
    assert(c.real() == -1.5);
    assert(c.imag() == 0);
  }

  {
    std::complex<T> c;
    const std::complex<T> c2(1.5, 2.5);
    assert(c.real() == 0);
    assert(c.imag() == 0);
    c -= c2;
    assert(c.real() == -1.5);
    assert(c.imag() == -2.5);
    c -= c2;
    assert(c.real() == -3);
    assert(c.imag() == -5);

    std::complex<T> c3;

    c3 = c;
    std::complex<int> ic (1,1);
    c3 -= ic;
    assert(c3.real() == -4);
    assert(c3.imag() == -6);

    c3 = c;
    std::complex<float> fc (1,1);
    c3 -= fc;
    assert(c3.real() == -4);
    assert(c3.imag() == -6);
  }
}

template <class T>
__device__ void test_times_equals() {
  {
    std::complex<T> c(1);
    c *= 1.5;
    c *= 1.5;
    c *= -1.5;
    c.imag(2);
    c *= 1.5;
    assert(c.real() == -5.0625);
    assert(c.imag() == 3);
  }

  {
    std::complex<T> c(1);
    const std::complex<T> c2(1.5, 2.5);
    c *= c2;
    c *= c2;
    assert(c.real() == -4);
    assert(c.imag() == 7.5);

    std::complex<T> c3;

    c3 = c;
    std::complex<int> ic (1,1);
    c3 *= ic;
    assert(c3.real() == -11.5);
    assert(c3.imag() ==   3.5);

    c3 = c;
    std::complex<float> fc (1,1);
    c3 *= fc;
    assert(c3.real() == -11.5);
    assert(c3.imag() ==   3.5);
  }
}

template <class T>
__device__ void test_divide_equals() {
  {
    std::complex<T> c(1);
    c /= 0.5;
    c /= 0.5;
    c /= -0.5;
    c.imag(2);
    c /= 0.5;
    assert(c.real() == -16);
    assert(c.imag() == 4);
  }

  {
    std::complex<T> c(-4, 7.5);
    const std::complex<T> c2(1.5, 2.5);
    assert(c.real() == -4);
    assert(c.imag() == 7.5);
    c /= c2;
    assert(c.real() == 1.5);
    assert(c.imag() == 2.5);
    c /= c2;
    assert(c.real() == 1);
    assert(c.imag() == 0);

    std::complex<T> c3;

    c3 = c;
    std::complex<int> ic (1,1);
    c3 /= ic;
    assert(c3.real() ==  0.5);
    assert(c3.imag() == -0.5);

    c3 = c;
    std::complex<float> fc (1,1);
    c3 /= fc;
    assert(c3.real() ==  0.5);
    assert(c3.imag() == -0.5);
  }
}

template <class T>
__device__ void test_construct() {
  {
    const std::complex<T> c;
    assert(c.real() == 0);
    assert(c.imag() == 0);
  }
  {
    const std::complex<T> c = 7.5;
    assert(c.real() == 7.5);
    assert(c.imag() == 0);
  }
  {
    const std::complex<T> c(8.5);
    assert(c.real() == 8.5);
    assert(c.imag() == 0);
  }
  {
    const std::complex<T> c(10.5, -9.5);
    assert(c.real() == 10.5);
    assert(c.imag() == -9.5);
  }
#if __cplusplus >= 201103L
  {
    constexpr std::complex<T> c;
    static_assert(c.real() == 0, "");
    static_assert(c.imag() == 0, "");
  }
  {
    constexpr std::complex<T> c = 7.5;
    static_assert(c.real() == 7.5, "");
    static_assert(c.imag() == 0, "");
  }
  {
    constexpr std::complex<T> c(8.5);
    static_assert(c.real() == 8.5, "");
    static_assert(c.imag() == 0, "");
  }
  {
    constexpr std::complex<T> c(10.5, -9.5);
    static_assert(c.real() == 10.5, "");
    static_assert(c.imag() == -9.5, "");
  }
#endif
}

template <class T>
__device__ void test_construct_integral() {
#if __cplusplus >= 201402L
  constexpr std::complex<T> c1;
  static_assert(c1.real() == 0, "");
  static_assert(c1.imag() == 0, "");
  constexpr std::complex<T> c2(3);
  static_assert(c2.real() == 3, "");
  static_assert(c2.imag() == 0, "");
  constexpr std::complex<T> c3(3, 4);
  static_assert(c3.real() == 3, "");
  static_assert(c3.imag() == 4, "");
#endif
}

template <class T>
__device__ void test_set_real_imag() {
  std::complex<T> c;
  c.real(3.5);
  assert(c.real() == 3.5);
  assert(c.imag() == 0);
  c.imag(4.5);
  assert(c.real() == 3.5);
  assert(c.imag() == 4.5);
}

template <class T>
__device__ void test_transcendentals_etc() {
  assert(sin(std::complex<T>(0, 0)) == std::complex<T>(0, 0));
  assert(sinh(std::complex<T>(0, 0)) == std::complex<T>(0, 0));
  assert(asin(std::complex<T>(0, 0)) == std::complex<T>(0, 0));
  assert(asinh(std::complex<T>(0, 0)) == std::complex<T>(0, 0));

  assert(cos(std::complex<T>(0, 0)) == std::complex<T>(1, 0));
  assert(cosh(std::complex<T>(0, 0)) == std::complex<T>(1, 0));
  {
    std::complex<T> c = acos(std::complex<T>(0, 0));
    is_about(real(c), T(M_PI_2));
    assert(std::abs(imag(c)) < 1.e-6);
  }
  {
    std::complex<T> c = acosh(std::complex<T>(0, 0));
    assert(std::abs(real(c)) < 1.e-6);
    is_about(imag(c), T(M_PI_2));
  }

  assert(tan(std::complex<T>(0, 0)) == std::complex<T>(0, 0));
  assert(tanh(std::complex<T>(0, 0)) == std::complex<T>(0, 0));
  assert(atan(std::complex<T>(0, 0)) == std::complex<T>(0, 0));
  assert(atanh(std::complex<T>(0, 0)) == std::complex<T>(0, 0));

  assert(exp(std::complex<T>(0, 0)) == std::complex<T>(1, 0));
  assert(log10(std::complex<T>(0, 0)) == std::complex<T>(-INFINITY, 0));
  assert(log(std::complex<T>(0, 0)) == std::complex<T>(-INFINITY, 0));

  {
    std::complex<T> c = pow(std::complex<T>(2, 3), std::complex<T>(2, 0));
    is_about(real(c), -5);
    is_about(imag(c), 12);
  }
  {
    std::complex<T> c = pow(std::complex<T>(2, 3), T(2));
    is_about(real(c), -5);
    is_about(imag(c), 12);
  }
  {
    std::complex<T> c = pow(T(2), std::complex<T>(2));
    is_about(real(c), 4);
    assert(std::abs(imag(c)) < 1.e-6);
  }
  {
    std::complex<T> c = sqrt(std::complex<T>(64, 0));
    is_about(real(c), 8);
    assert(std::abs(imag(c)) < 1.e-6);
  }

  // "etc."
  assert(abs(std::complex<T>(3, 4)) == 5);
  assert(norm(std::complex<T>(3, 4)) == 25);
  assert(arg(std::complex<T>(1, 0)) == 0);
  assert(conj(std::complex<T>(1, 2)) == std::complex<T>(1, -2));

  assert(std::polar(T(0)) == std::complex<T>(0, 0));
  assert(std::polar(T(1)) == std::complex<T>(1, 0));
  assert(std::polar(T(100)) == std::complex<T>(100, 0));
  assert(std::polar(T(0), T(0)) == std::complex<T>(0, 0));
  assert(std::polar(T(1), T(0)) == std::complex<T>(1, 0));
  assert(std::polar(T(100), T(0)) == std::complex<T>(100, 0));

#ifndef __GLIBCXX__
  // libstdc++'s implementation of proj is completely broken, see
  // https://gcc.gnu.org/bugzilla/show_bug.cgi?id=61761.
  assert(std::proj(std::complex<T>(1, 2)) == std::complex<T>(1, 2));
  assert(std::proj(std::complex<T>(-1, 2)) == std::complex<T>(-1, 2));
  assert(std::proj(std::complex<T>(1, -2)) == std::complex<T>(1, -2));
  assert(std::proj(std::complex<T>(-1, -2)) == std::complex<T>(-1, -2));
#endif
}

__global__ void tests() {
  test_promotion();
  test_literals();
  test_assignment_real<float>();
  test_assignment_real<double>();

  test_assignment_complex<float, float>();
  test_assignment_complex<float, double>();
  test_assignment_complex<double, float>();
  test_assignment_complex<double, double>();

  test_plus_equals<float>();
  test_plus_equals<double>();
  test_minus_equals<float>();
  test_minus_equals<double>();
  test_times_equals<float>();
  test_times_equals<double>();
  test_divide_equals<float>();
  test_divide_equals<double>();

  test_construct<float>();
  test_construct<double>();
  test_construct_integral<int>();

  test_set_real_imag<float>();
  test_set_real_imag<double>();

  test_transcendentals_etc<float>();
  test_transcendentals_etc<double>();
}
#else
__global__ void tests() {}
#endif

int main() {
  tests<<<1, 1>>>();
  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA error %d\n", (int)err);
    return 1;
  }
  printf("Success!\n");
  return 0;
}
