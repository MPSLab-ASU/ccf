// Make sure that we can compile CUDA files that include <future> and use
// std::shared_future.
//
// At one point in time this didn't work because clang defined different values
// for __GCC_ATOMIC_INT_LOCK_FREE on host and device.  This caused libstdc++
// not to define std::shared_future when compiling for device, resulting in
// compile errors (even though we only use std::shared_future in host code).

#if __cplusplus >= 201103L

#include <hip/hip_runtime.h>
#include <future>

void foo() {
  std::shared_future<int> x;
}
#else
#warning Skipping test because not compiled in C++11 mode.
#endif

int main() {
  return 0;
}
