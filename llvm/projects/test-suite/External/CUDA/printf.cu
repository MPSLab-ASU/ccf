
#include <hip/hip_runtime.h>
__global__ void kernel(char a, short b, int c, float d, double e) {
  const char* f = "abcdefg";
  printf("kernel_printf %c %d %d %f %f %s\n", a, b, c, d, e, f);
}

int main(int argc, char* argv[]) {
  kernel<<<2, 8>>>('a', 2, 3, 4, 5);
  hipDeviceSynchronize();
  hipDeviceReset();
  return 0;
}
