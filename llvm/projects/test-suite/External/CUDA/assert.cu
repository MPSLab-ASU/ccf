
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>

// Use fixed value for __FILE__ so assert message does not depend on
// the actual path to the file during compilation..
#define __FILE__ "assert.cu"

__global__ void kernel() {
  // Our reference output contains the line number of this assert() call; be
  // careful when modifying the parts of this file above this line.
  assert(false);
}

int main() {
  kernel<<<1,1>>>();
  hipError_t err = hipDeviceSynchronize();
  if (err != hipErrorAssert)
    return err;
  return 0;
}
