#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
//                     The LLVM Compiler Infrastructure
//
// This file is dual licensed under the MIT and the University of Illinois Open
// Source Licenses. See LICENSE.TXT for details.
//
//===----------------------------------------------------------------------===//

// This file was copied from libc++'s test suite, then modified to test CUDA.
// For the most part, this consists of adding __device__ attributes and
// deleting long double.

// <cmath>

// This test requires c++11 (it's mostly decltype stuff).

#if __cplusplus >= 201103L

#include <cmath>
#include <type_traits>
#include <cassert>
#include <stdio.h>

// See PR21083
// Ambiguous is a user-defined type that defines its own overloads of cmath
// functions. When the std overloads are candidates too (by using or adl),
// they should not interfere.
struct Ambiguous : std::true_type { // ADL
    __device__ operator float () { return 0.f; }
    __device__ operator double () { return 0.; }
};
__device__ Ambiguous abs(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous acos(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous asin(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous atan(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous atan2(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous ceil(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous cos(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous cosh(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous exp(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous fabs(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous floor(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous fmod(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous frexp(Ambiguous, int*){ return Ambiguous(); }
__device__ Ambiguous ldexp(Ambiguous, int){ return Ambiguous(); }
__device__ Ambiguous log(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous log10(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous modf(Ambiguous, Ambiguous*){ return Ambiguous(); }
__device__ Ambiguous pow(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous sin(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous sinh(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous sqrt(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous tan(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous tanh(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous signbit(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous fpclassify(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous isfinite(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous isnormal(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous isgreater(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous isgreaterequal(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous isless(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous islessequal(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous islessgreater(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous isunordered(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous acosh(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous asinh(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous atanh(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous cbrt(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous copysign(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous erf(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous erfc(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous exp2(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous expm1(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous fdim(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous fma(Ambiguous, Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous fmax(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous fmin(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous hypot(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous hypot(Ambiguous, Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous ilogb(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous lgamma(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous llrint(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous llround(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous log1p(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous log2(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous logb(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous lrint(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous lround(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous nearbyint(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous nextafter(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous nexttoward(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous remainder(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous remquo(Ambiguous, Ambiguous, int*){ return Ambiguous(); }
__device__ Ambiguous rint(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous round(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous scalbln(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous scalbn(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous tgamma(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous trunc(Ambiguous){ return Ambiguous(); }

__device__ void test_abs()
{
    static_assert((std::is_same<decltype(std::abs((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::abs((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(abs(Ambiguous())), Ambiguous>::value), "");
    assert(std::abs(-1) == 1);
    assert(std::abs(-1.) == 1);
    assert(std::abs(-1.f) == 1);
}

__device__ void test_acos()
{
    static_assert((std::is_same<decltype(std::acos((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::acos((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::acos((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::acos((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::acos((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::acos((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::acos((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::acos((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::acos((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::acos((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::acosf(0)), float>::value), "");
    static_assert((std::is_same<decltype(acos(Ambiguous())), Ambiguous>::value), "");
    assert(std::acos(1) == 0);
    assert(std::acos(1.) == 0);
    assert(std::acos(1.f) == 0);
}

__device__ void test_asin()
{
    static_assert((std::is_same<decltype(std::asin((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::asin((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::asin((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::asin((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::asin((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::asin((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::asin((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::asin((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::asin((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::asin((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::asinf(0)), float>::value), "");
    static_assert((std::is_same<decltype(asin(Ambiguous())), Ambiguous>::value), "");
    assert(std::asin(0) == 0);
    assert(std::asin(0.) == 0);
    assert(std::asin(0.f) == 0);
}

__device__ void test_atan()
{
    static_assert((std::is_same<decltype(std::atan((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::atan((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::atan((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::atan((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::atan((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::atan((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::atan((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::atan((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::atan((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::atan((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::atanf(0)), float>::value), "");
    static_assert((std::is_same<decltype(atan(Ambiguous())), Ambiguous>::value), "");
    assert(std::atan(0) == 0);
    assert(std::atan(0.) == 0);
    assert(std::atan(0.f) == 0);
}

__device__ void test_atan2()
{
    static_assert((std::is_same<decltype(std::atan2((float)0, (float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::atan2((bool)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::atan2((unsigned short)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::atan2((float)0, (unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::atan2((double)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::atan2((int)0, (long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::atan2((int)0, (unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::atan2((double)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::atan2((float)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::atan2f(0,0)), float>::value), "");
    static_assert((std::is_same<decltype(std::atan2((int)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(atan2(Ambiguous(), Ambiguous())), Ambiguous>::value), "");

    assert(std::atan2(0, 1) == 0);
    assert(std::atan2(0, 1.) == 0);
    assert(std::atan2(0, 1.f) == 0);

    assert(std::atan2(0., 1) == 0);
    assert(std::atan2(0., 1.) == 0);
    assert(std::atan2(0., 1.f) == 0);

    assert(std::atan2(0.f, 1) == 0);
    assert(std::atan2(0.f, 1.) == 0);
    assert(std::atan2(0.f, 1.f) == 0);
}

__device__ void test_ceil()
{
    static_assert((std::is_same<decltype(std::ceil((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::ceil((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::ceil((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::ceil((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::ceil((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::ceil((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::ceil((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::ceil((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::ceil((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::ceil((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::ceilf(0)), float>::value), "");
    static_assert((std::is_same<decltype(ceil(Ambiguous())), Ambiguous>::value), "");
    assert(std::ceil(0) == 0);
    assert(std::ceil(0.) == 0);
    assert(std::ceil(0.f) == 0);
}

__device__ void test_cos()
{
    static_assert((std::is_same<decltype(std::cos((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::cos((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::cos((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::cos((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::cos((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::cos((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::cos((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::cos((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::cos((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::cos((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::cosf(0)), float>::value), "");
    static_assert((std::is_same<decltype(cos(Ambiguous())), Ambiguous>::value), "");
    assert(std::cos(0) == 1);
    assert(std::cos(0.) == 1);
    assert(std::cos(0.f) == 1);
}

__device__ void test_cosh()
{
    static_assert((std::is_same<decltype(std::cosh((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::cosh((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::cosh((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::cosh((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::cosh((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::cosh((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::cosh((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::cosh((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::cosh((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::cosh((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::coshf(0)), float>::value), "");
    static_assert((std::is_same<decltype(cosh(Ambiguous())), Ambiguous>::value), "");
    assert(std::cosh(0) == 1);
    assert(std::cosh(0.) == 1);
    assert(std::cosh(0.f) == 1);
}

__device__ void test_exp()
{
    static_assert((std::is_same<decltype(std::exp((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::exp((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::exp((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::exp((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::exp((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::exp((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::exp((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::exp((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::exp((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::exp((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::expf(0)), float>::value), "");
    static_assert((std::is_same<decltype(exp(Ambiguous())), Ambiguous>::value), "");
    assert(std::exp(0) == 1);
    assert(std::exp(0.) == 1);
    assert(std::exp(0.f) == 1);
}

__device__ void test_fabs()
{
    static_assert((std::is_same<decltype(std::fabs((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::fabs((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fabs((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fabs((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fabs((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fabs((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fabs((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fabs((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fabs((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fabs((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fabsf(0.0f)), float>::value), "");
    static_assert((std::is_same<decltype(fabs(Ambiguous())), Ambiguous>::value), "");
    assert(std::fabs(-1) == 1);
    assert(std::fabs(-1.) == 1);
    assert(std::fabs(-1.f) == 1);
}

__device__ void test_floor()
{
    static_assert((std::is_same<decltype(std::floor((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::floor((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::floor((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::floor((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::floor((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::floor((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::floor((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::floor((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::floor((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::floor((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::floorf(0)), float>::value), "");
    static_assert((std::is_same<decltype(floor(Ambiguous())), Ambiguous>::value), "");
    assert(std::floor(1) == 1);
    assert(std::floor(1.) == 1);
    assert(std::floor(1.f) == 1);
}

__device__ void test_fmod()
{
    static_assert((std::is_same<decltype(std::fmod((float)0, (float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::fmod((bool)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fmod((unsigned short)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fmod((float)0, (unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fmod((double)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fmod((int)0, (long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fmod((int)0, (unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fmod((double)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fmod((float)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fmodf(0,0)), float>::value), "");
    static_assert((std::is_same<decltype(std::fmod((int)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(fmod(Ambiguous(), Ambiguous())), Ambiguous>::value), "");

    assert(std::fmod(1.5, 1) == .5);
    assert(std::fmod(1.5, 1.) == .5);
    assert(std::fmod(1.5, 1.f) == .5);

    assert(std::fmod(1.5f, 1) == .5);
    assert(std::fmod(1.5f, 1.) == .5);
    assert(std::fmod(1.5f, 1.f) == .5);

    assert(std::fmod(2, 1) == 0);
    assert(std::fmod(2, 1.) == 0);
    assert(std::fmod(2, 1.f) == 0);
}

__device__ void test_frexp()
{
    int ip;
    static_assert((std::is_same<decltype(std::frexp((float)0, &ip)), float>::value), "");
    static_assert((std::is_same<decltype(std::frexp((bool)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(std::frexp((unsigned short)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(std::frexp((int)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(std::frexp((unsigned int)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(std::frexp((long)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(std::frexp((unsigned long)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(std::frexp((long long)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(std::frexp((unsigned long long)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(std::frexp((double)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(std::frexpf(0, &ip)), float>::value), "");
    static_assert((std::is_same<decltype(frexp(Ambiguous(), &ip)), Ambiguous>::value), "");
    assert(std::frexp(0, &ip) == 0);
    assert(std::frexp(0., &ip) == 0);
    assert(std::frexp(0.f, &ip) == 0);
}

__device__ void test_ldexp()
{
    int ip = 1;
    static_assert((std::is_same<decltype(std::ldexp((float)0, ip)), float>::value), "");
    static_assert((std::is_same<decltype(std::ldexp((bool)0, ip)), double>::value), "");
    static_assert((std::is_same<decltype(std::ldexp((unsigned short)0, ip)), double>::value), "");
    static_assert((std::is_same<decltype(std::ldexp((int)0, ip)), double>::value), "");
    static_assert((std::is_same<decltype(std::ldexp((unsigned int)0, ip)), double>::value), "");
    static_assert((std::is_same<decltype(std::ldexp((long)0, ip)), double>::value), "");
    static_assert((std::is_same<decltype(std::ldexp((unsigned long)0, ip)), double>::value), "");
    static_assert((std::is_same<decltype(std::ldexp((long long)0, ip)), double>::value), "");
    static_assert((std::is_same<decltype(std::ldexp((unsigned long long)0, ip)), double>::value), "");
    static_assert((std::is_same<decltype(std::ldexp((double)0, ip)), double>::value), "");
    static_assert((std::is_same<decltype(std::ldexpf(0, ip)), float>::value), "");
    static_assert((std::is_same<decltype(ldexp(Ambiguous(), ip)), Ambiguous>::value), "");
    assert(std::ldexp(1, ip) == 2);
    assert(std::ldexp(1., ip) == 2);
    assert(std::ldexp(1.f, ip) == 2);
}

__device__ void test_log()
{
    static_assert((std::is_same<decltype(std::log((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::log((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::logf(0)), float>::value), "");
    static_assert((std::is_same<decltype(log(Ambiguous())), Ambiguous>::value), "");
    assert(std::log(1) == 0);
    assert(std::log(1.) == 0);
    assert(std::log(1.f) == 0);
}

__device__ void test_log10()
{
    static_assert((std::is_same<decltype(std::log10((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::log10((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log10((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log10((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log10((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log10((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log10((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log10((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log10((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log10((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log10f(0)), float>::value), "");
    static_assert((std::is_same<decltype(log10(Ambiguous())), Ambiguous>::value), "");
    assert(std::log10(1) == 0);
    assert(std::log10(1.) == 0);
    assert(std::log10(1.f) == 0);
}

__device__ void test_modf()
{
    static_assert((std::is_same<decltype(std::modf((float)0, (float*)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::modf((double)0, (double*)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::modff(0, (float*)0)), float>::value), "");
    static_assert((std::is_same<decltype(modf(Ambiguous(), (Ambiguous*)0)), Ambiguous>::value), "");
    double i;
    assert(std::modf(1, &i) == 0);
    assert(std::modf(1., &i) == 0);
    assert(std::modf(1.f, &i) == 0);
}

__device__ void test_pow()
{
    static_assert((std::is_same<decltype(std::pow((float)0, (float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::pow((bool)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::pow((unsigned short)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::pow((float)0, (unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::pow((double)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::pow((int)0, (long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::pow((int)0, (unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::pow((double)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::pow((float)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::powf(0,0)), float>::value), "");
    static_assert((std::is_same<decltype(std::pow((int)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(pow(Ambiguous(), Ambiguous())), Ambiguous>::value), "");
    assert(std::pow(1, 1) == 1);
    assert(std::pow(1., 1) == 1);
    assert(std::pow(1.f, 1) == 1);

    assert(std::pow(1, 1.) == 1);
    assert(std::pow(1., 1.) == 1);
    assert(std::pow(1.f, 1.) == 1);

    assert(std::pow(1, 1.f) == 1);
    assert(std::pow(1., 1.f) == 1);
    assert(std::pow(1.f, 1.f) == 1);
}

__device__ void test_sin()
{
    static_assert((std::is_same<decltype(std::sin((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::sin((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::sin((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::sin((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::sin((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::sin((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::sin((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::sin((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::sin((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::sin((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::sinf(0)), float>::value), "");
    static_assert((std::is_same<decltype(sin(Ambiguous())), Ambiguous>::value), "");
    assert(std::sin(0) == 0);
    assert(std::sin(0.) == 0);
    assert(std::sin(0.f) == 0);
}

__device__ void test_sinh()
{
    static_assert((std::is_same<decltype(std::sinh((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::sinh((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::sinh((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::sinh((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::sinh((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::sinh((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::sinh((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::sinh((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::sinh((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::sinh((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::sinhf(0)), float>::value), "");
    static_assert((std::is_same<decltype(sinh(Ambiguous())), Ambiguous>::value), "");
    assert(std::sinh(0) == 0);
    assert(std::sinh(0.) == 0);
    assert(std::sinh(0.f) == 0);
}

__device__ void test_sqrt()
{
    static_assert((std::is_same<decltype(std::sqrt((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::sqrt((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::sqrt((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::sqrt((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::sqrt((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::sqrt((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::sqrt((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::sqrt((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::sqrt((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::sqrt((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::sqrtf(0)), float>::value), "");
    static_assert((std::is_same<decltype(sqrt(Ambiguous())), Ambiguous>::value), "");
    assert(std::sqrt(4) == 2);
    assert(std::sqrt(4.) == 2);
    assert(std::sqrt(4.f) == 2);
}

__device__ void test_tan()
{
    static_assert((std::is_same<decltype(std::tan((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::tan((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::tan((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::tan((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::tan((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::tan((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::tan((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::tan((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::tan((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::tan((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::tanf(0)), float>::value), "");
    static_assert((std::is_same<decltype(tan(Ambiguous())), Ambiguous>::value), "");
    assert(std::tan(0) == 0);
    assert(std::tan(0.) == 0);
    assert(std::tan(0.f) == 0);
}

__device__ void test_tanh()
{
    static_assert((std::is_same<decltype(std::tanh((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::tanh((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::tanh((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::tanh((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::tanh((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::tanh((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::tanh((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::tanh((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::tanh((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::tanh((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::tanhf(0)), float>::value), "");
    static_assert((std::is_same<decltype(tanh(Ambiguous())), Ambiguous>::value), "");
    assert(std::tanh(0) == 0);
    assert(std::tanh(0.) == 0);
    assert(std::tanh(0.f) == 0);
}

__device__ void test_signbit()
{
#ifdef signbit
#error signbit defined
#endif
    static_assert((std::is_same<decltype(std::signbit((float)0)), bool>::value), "");
    static_assert((std::is_same<decltype(std::signbit((double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(std::signbit(0)), bool>::value), "");
    static_assert((std::is_same<decltype(signbit(Ambiguous())), Ambiguous>::value), "");
    assert(std::signbit(-1) == true);
    assert(std::signbit(-1.) == true);
    assert(std::signbit(-1.f) == true);
}

__device__ void test_fpclassify()
{
#ifdef fpclassify
#error fpclassify defined
#endif
    static_assert((std::is_same<decltype(std::fpclassify((float)0)), int>::value), "");
    static_assert((std::is_same<decltype(std::fpclassify((double)0)), int>::value), "");
    static_assert((std::is_same<decltype(std::fpclassify(0)), int>::value), "");
    static_assert((std::is_same<decltype(fpclassify(Ambiguous())), Ambiguous>::value), "");
    assert(std::fpclassify(-1) == FP_NORMAL);
    assert(std::fpclassify(-1.) == FP_NORMAL);
    assert(std::fpclassify(-1.f) == FP_NORMAL);
}

__device__ void test_isfinite()
{
#ifdef isfinite
#error isfinite defined
#endif
    static_assert((std::is_same<decltype(std::isfinite((float)0)), bool>::value), "");
    static_assert((std::is_same<decltype(std::isfinite((double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(std::isfinite(0)), bool>::value), "");
    static_assert((std::is_same<decltype(isfinite(Ambiguous())), Ambiguous>::value), "");
    assert(std::isfinite(-1) == true);
    assert(std::isfinite(-1.) == true);
    assert(std::isfinite(-1.f) == true);
}

__device__ void test_isnormal()
{
#ifdef isnormal
#error isnormal defined
#endif
    static_assert((std::is_same<decltype(std::isnormal((float)0)), bool>::value), "");
    static_assert((std::is_same<decltype(std::isnormal((double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(std::isnormal(0)), bool>::value), "");
    static_assert((std::is_same<decltype(isnormal(Ambiguous())), Ambiguous>::value), "");
    assert(std::isnormal(-1) == true);
    assert(std::isnormal(-1.) == true);
    assert(std::isnormal(-1.f) == true);
}

__device__ void test_isgreater()
{
#ifdef isgreater
#error isgreater defined
#endif
    static_assert((std::is_same<decltype(std::isgreater((float)0, (float)0)), bool>::value), "");
    static_assert((std::is_same<decltype(std::isgreater((float)0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(std::isgreater((double)0, (float)0)), bool>::value), "");
    static_assert((std::is_same<decltype(std::isgreater((double)0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(std::isgreater(0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(isgreater(Ambiguous(), Ambiguous())), Ambiguous>::value), "");
    assert(std::isgreater(-1, 0) == false);
    assert(std::isgreater(-1, 0.) == false);
    assert(std::isgreater(-1, 0.f) == false);

    assert(std::isgreater(-1., 0) == false);
    assert(std::isgreater(-1., 0.) == false);
    assert(std::isgreater(-1., 0.f) == false);

    assert(std::isgreater(-1.f, 0) == false);
    assert(std::isgreater(-1.f, 0.) == false);
    assert(std::isgreater(-1.f, 0.f) == false);
}

__device__ void test_isgreaterequal()
{
#ifdef isgreaterequal
#error isgreaterequal defined
#endif
    static_assert((std::is_same<decltype(std::isgreaterequal((float)0, (float)0)), bool>::value), "");
    static_assert((std::is_same<decltype(std::isgreaterequal((float)0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(std::isgreaterequal((double)0, (float)0)), bool>::value), "");
    static_assert((std::is_same<decltype(std::isgreaterequal((double)0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(std::isgreaterequal(0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(isgreaterequal(Ambiguous(), Ambiguous())), Ambiguous>::value), "");
    assert(std::isgreaterequal(-1, 0) == false);
    assert(std::isgreaterequal(-1, 0.) == false);
    assert(std::isgreaterequal(-1, 0.f) == false);

    assert(std::isgreaterequal(-1., 0) == false);
    assert(std::isgreaterequal(-1., 0.) == false);
    assert(std::isgreaterequal(-1., 0.f) == false);

    assert(std::isgreaterequal(-1.f, 0) == false);
    assert(std::isgreaterequal(-1.f, 0.) == false);
    assert(std::isgreaterequal(-1.f, 0.f) == false);
}

__device__ void test_isinf()
{
#ifdef isinf
#error isinf defined
#endif
    static_assert((std::is_same<decltype(std::isinf((float)0)), bool>::value), "");

    typedef decltype(std::isinf((double)0)) DoubleRetType;
#ifndef __linux__
    static_assert((std::is_same<DoubleRetType, bool>::value), "");
#else
    // GLIBC < 2.26 defines 'isinf(double)' with a return type of 'int' in
    // all C++ dialects. The test should tolerate this.
    // See: https://sourceware.org/bugzilla/show_bug.cgi?id=19439
    static_assert((std::is_same<DoubleRetType, bool>::value
                || std::is_same<DoubleRetType, int>::value), "");
#endif

    static_assert((std::is_same<decltype(std::isinf(0)), bool>::value), "");
    assert(std::isinf(-1) == false);
    assert(std::isinf(-1.) == false);
    assert(std::isinf(-1.f) == false);
}

__device__ void test_isless()
{
#ifdef isless
#error isless defined
#endif
    static_assert((std::is_same<decltype(std::isless((float)0, (float)0)), bool>::value), "");
    static_assert((std::is_same<decltype(std::isless((float)0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(std::isless((double)0, (float)0)), bool>::value), "");
    static_assert((std::is_same<decltype(std::isless((double)0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(std::isless(0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(isless(Ambiguous(), Ambiguous())), Ambiguous>::value), "");
    assert(std::isless(-1, 0) == true);
    assert(std::isless(-1, 0.) == true);
    assert(std::isless(-1, 0.f) == true);

    assert(std::isless(-1., 0) == true);
    assert(std::isless(-1., 0.) == true);
    assert(std::isless(-1., 0.f) == true);

    assert(std::isless(-1.f, 0) == true);
    assert(std::isless(-1.f, 0.) == true);
    assert(std::isless(-1.f, 0.f) == true);
}

__device__ void test_islessequal()
{
#ifdef islessequal
#error islessequal defined
#endif
    static_assert((std::is_same<decltype(std::islessequal((float)0, (float)0)), bool>::value), "");
    static_assert((std::is_same<decltype(std::islessequal((float)0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(std::islessequal((double)0, (float)0)), bool>::value), "");
    static_assert((std::is_same<decltype(std::islessequal((double)0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(std::islessequal(0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(islessequal(Ambiguous(), Ambiguous())), Ambiguous>::value), "");
    assert(std::islessequal(-1, 0) == true);
    assert(std::islessequal(-1, 0.) == true);
    assert(std::islessequal(-1, 0.f) == true);

    assert(std::islessequal(-1., 0) == true);
    assert(std::islessequal(-1., 0.) == true);
    assert(std::islessequal(-1., 0.f) == true);

    assert(std::islessequal(-1.f, 0) == true);
    assert(std::islessequal(-1.f, 0.) == true);
    assert(std::islessequal(-1.f, 0.f) == true);
}

__device__ void test_islessgreater()
{
#ifdef islessgreater
#error islessgreater defined
#endif
    static_assert((std::is_same<decltype(std::islessgreater((float)0, (float)0)), bool>::value), "");
    static_assert((std::is_same<decltype(std::islessgreater((float)0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(std::islessgreater((double)0, (float)0)), bool>::value), "");
    static_assert((std::is_same<decltype(std::islessgreater((double)0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(std::islessgreater(0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(islessgreater(Ambiguous(), Ambiguous())), Ambiguous>::value), "");
    assert(std::islessgreater(-1, 0) == true);
    assert(std::islessgreater(-1, 0.) == true);
    assert(std::islessgreater(-1, 0.f) == true);

    assert(std::islessgreater(-1., 0) == true);
    assert(std::islessgreater(-1., 0.) == true);
    assert(std::islessgreater(-1., 0.f) == true);

    assert(std::islessgreater(-1.f, 0) == true);
    assert(std::islessgreater(-1.f, 0.) == true);
    assert(std::islessgreater(-1.f, 0.f) == true);
}

__device__ void test_isnan()
{
#ifdef isnan
#error isnan defined
#endif
    static_assert((std::is_same<decltype(std::isnan((float)0)), bool>::value), "");

    typedef decltype(std::isnan((double)0)) DoubleRetType;
#ifndef __linux__
    static_assert((std::is_same<DoubleRetType, bool>::value), "");
#else
    // GLIBC < 2.26 defines 'isnan(double)' with a return type of 'int' in
    // all C++ dialects. The test should tolerate this.
    // See: https://sourceware.org/bugzilla/show_bug.cgi?id=19439
    static_assert((std::is_same<DoubleRetType, bool>::value
                || std::is_same<DoubleRetType, int>::value), "");
#endif

    static_assert((std::is_same<decltype(std::isnan(0)), bool>::value), "");
    assert(std::isnan(-1) == false);
    assert(std::isnan(-1.) == false);
    assert(std::isnan(-1.f) == false);
}

__device__ void test_isunordered()
{
#ifdef isunordered
#error isunordered defined
#endif
    static_assert((std::is_same<decltype(std::isunordered((float)0, (float)0)), bool>::value), "");
    static_assert((std::is_same<decltype(std::isunordered((float)0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(std::isunordered((double)0, (float)0)), bool>::value), "");
    static_assert((std::is_same<decltype(std::isunordered((double)0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(std::isunordered(0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(isunordered(Ambiguous(), Ambiguous())), Ambiguous>::value), "");
    assert(std::isunordered(-1, 0) == false);
    assert(std::isunordered(-1, 0.) == false);
    assert(std::isunordered(-1, 0.f) == false);

    assert(std::isunordered(-1., 0) == false);
    assert(std::isunordered(-1., 0.) == false);
    assert(std::isunordered(-1., 0.f) == false);

    assert(std::isunordered(-1.f, 0) == false);
    assert(std::isunordered(-1.f, 0.) == false);
    assert(std::isunordered(-1.f, 0.f) == false);
}

__device__ void test_acosh()
{
    static_assert((std::is_same<decltype(std::acosh((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::acosh((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::acosh((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::acosh((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::acosh((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::acosh((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::acosh((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::acosh((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::acosh((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::acosh((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::acoshf(0)), float>::value), "");
    static_assert((std::is_same<decltype(acosh(Ambiguous())), Ambiguous>::value), "");
    assert(std::acosh(1) == 0);
    assert(std::acosh(1.) == 0);
    assert(std::acosh(1.f) == 0);
}

__device__ void test_asinh()
{
    static_assert((std::is_same<decltype(std::asinh((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::asinh((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::asinh((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::asinh((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::asinh((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::asinh((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::asinh((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::asinh((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::asinh((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::asinh((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::asinhf(0)), float>::value), "");
    static_assert((std::is_same<decltype(asinh(Ambiguous())), Ambiguous>::value), "");
    assert(std::asinh(0) == 0);
    assert(std::asinh(0.) == 0);
    assert(std::asinh(0.f) == 0);
}

__device__ void test_atanh()
{
    static_assert((std::is_same<decltype(std::atanh((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::atanh((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::atanh((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::atanh((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::atanh((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::atanh((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::atanh((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::atanh((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::atanh((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::atanh((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::atanhf(0)), float>::value), "");
    static_assert((std::is_same<decltype(atanh(Ambiguous())), Ambiguous>::value), "");
    assert(std::atanh(0) == 0);
    assert(std::atanh(0.) == 0);
    assert(std::atanh(0.f) == 0);
}

__device__ void test_cbrt()
{
    static_assert((std::is_same<decltype(std::cbrt((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::cbrt((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::cbrt((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::cbrt((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::cbrt((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::cbrt((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::cbrt((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::cbrt((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::cbrt((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::cbrt((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::cbrtf(0)), float>::value), "");
    static_assert((std::is_same<decltype(cbrt(Ambiguous())), Ambiguous>::value), "");
    assert(std::cbrt(1) == 1);
    assert(std::cbrt(1.) == 1);
    assert(std::cbrt(1.f) == 1);
}

__device__ void test_copysign()
{
    static_assert((std::is_same<decltype(std::copysign((float)0, (float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::copysign((bool)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::copysign((unsigned short)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::copysign((float)0, (unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::copysign((double)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::copysign((int)0, (long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::copysign((int)0, (unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::copysign((double)0, (double)0)), double>::value), "");
    // CUDA's copysign(float, double) returns a float.  This is not per spec,
    // but it's kind of reasonable -- given that copysign just copies the sign
    // of the LHS to the RHS, there's no reason that we should have to promote
    // the LHS from float to double.
    //static_assert((std::is_same<decltype(std::copysign((float)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::copysignf(0,0)), float>::value), "");
    static_assert((std::is_same<decltype(std::copysign((int)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(copysign(Ambiguous(), Ambiguous())), Ambiguous>::value), "");

    assert(std::copysign(1, 1) == 1);
    assert(std::copysign(1., 1) == 1);
    assert(std::copysign(1.f, 1) == 1);

    assert(std::copysign(1, 1.) == 1);
    assert(std::copysign(1., 1.) == 1);
    assert(std::copysign(1.f, 1.) == 1);

    assert(std::copysign(1, 1.f) == 1);
    assert(std::copysign(1., 1.f) == 1);
    assert(std::copysign(1.f, 1.f) == 1);
}

__device__ void test_erf()
{
    static_assert((std::is_same<decltype(std::erf((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::erf((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::erf((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::erf((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::erf((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::erf((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::erf((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::erf((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::erf((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::erf((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::erff(0)), float>::value), "");
    static_assert((std::is_same<decltype(erf(Ambiguous())), Ambiguous>::value), "");
    assert(std::erf(0) == 0);
    assert(std::erf(0.) == 0);
    assert(std::erf(0.f) == 0);
}

__device__ void test_erfc()
{
    static_assert((std::is_same<decltype(std::erfc((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::erfc((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::erfc((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::erfc((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::erfc((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::erfc((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::erfc((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::erfc((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::erfc((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::erfc((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::erfcf(0)), float>::value), "");
    static_assert((std::is_same<decltype(erfc(Ambiguous())), Ambiguous>::value), "");
    assert(std::erfc(0) == 1);
    assert(std::erfc(0.) == 1);
    assert(std::erfc(0.f) == 1);
}

__device__ void test_exp2()
{
    static_assert((std::is_same<decltype(std::exp2((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::exp2((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::exp2((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::exp2((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::exp2((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::exp2((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::exp2((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::exp2((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::exp2((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::exp2((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::exp2f(0)), float>::value), "");
    static_assert((std::is_same<decltype(exp2(Ambiguous())), Ambiguous>::value), "");
    assert(std::exp2(1) == 2);
    assert(std::exp2(1.) == 2);
    assert(std::exp2(1.f) == 2);
}

__device__ void test_expm1()
{
    static_assert((std::is_same<decltype(std::expm1((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::expm1((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::expm1((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::expm1((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::expm1((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::expm1((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::expm1((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::expm1((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::expm1((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::expm1((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::expm1f(0)), float>::value), "");
    static_assert((std::is_same<decltype(expm1(Ambiguous())), Ambiguous>::value), "");
    assert(std::expm1(0) == 0);
    assert(std::expm1(0.) == 0);
    assert(std::expm1(0.f) == 0);
}

__device__ void test_fdim()
{
    static_assert((std::is_same<decltype(std::fdim((float)0, (float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::fdim((bool)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fdim((unsigned short)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fdim((float)0, (unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fdim((double)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fdim((int)0, (long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fdim((int)0, (unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fdim((double)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fdim((float)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fdimf(0,0)), float>::value), "");
    static_assert((std::is_same<decltype(std::fdim((int)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(fdim(Ambiguous(), Ambiguous())), Ambiguous>::value), "");

    assert(std::fdim(1, 0) == 1);
    assert(std::fdim(1., 0) == 1);
    assert(std::fdim(1.f, 0) == 1);

    assert(std::fdim(1, 0.) == 1);
    assert(std::fdim(1., 0.) == 1);
    assert(std::fdim(1.f, 0.) == 1);

    assert(std::fdim(1, 0.f) == 1);
    assert(std::fdim(1., 0.f) == 1);
    assert(std::fdim(1.f, 0.f) == 1);
}

__device__ void test_fma()
{
    static_assert((std::is_same<decltype(std::fma((bool)0, (float)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fma((char)0, (float)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fma((unsigned)0, (float)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fma((float)0, (int)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fma((float)0, (long)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fma((float)0, (float)0, (unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fma((float)0, (float)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fma((float)0, (float)0, (float)0)), float>::value), "");

    static_assert((std::is_same<decltype(std::fma((bool)0, (double)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fma((char)0, (double)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fma((unsigned)0, (double)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fma((double)0, (int)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fma((double)0, (long)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fma((double)0, (double)0, (unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fma((double)0, (double)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fma((double)0, (double)0,  (double)0)), double>::value), "");


    static_assert((std::is_same<decltype(std::fmaf(0,0,0)), float>::value), "");
    static_assert((std::is_same<decltype(fma(Ambiguous(), Ambiguous(), Ambiguous())), Ambiguous>::value), "");

    assert(std::fma(1, 1, 1) == 2);
    assert(std::fma(1., 1, 1) == 2);
    assert(std::fma(1.f, 1, 1) == 2);
    assert(std::fma(1, 1., 1) == 2);
    assert(std::fma(1., 1., 1) == 2);
    assert(std::fma(1.f, 1., 1) == 2);
    assert(std::fma(1, 1.f, 1) == 2);
    assert(std::fma(1., 1.f, 1) == 2);
    assert(std::fma(1.f, 1.f, 1) == 2);

    assert(std::fma(1, 1, 1.) == 2);
    assert(std::fma(1., 1, 1.) == 2);
    assert(std::fma(1.f, 1, 1.) == 2);
    assert(std::fma(1, 1., 1.) == 2);
    assert(std::fma(1., 1., 1.) == 2);
    assert(std::fma(1.f, 1., 1.) == 2);
    assert(std::fma(1, 1.f, 1.) == 2);
    assert(std::fma(1., 1.f, 1.) == 2);
    assert(std::fma(1.f, 1.f, 1.) == 2);

    assert(std::fma(1, 1, 1.f) == 2);
    assert(std::fma(1., 1, 1.f) == 2);
    assert(std::fma(1.f, 1, 1.f) == 2);
    assert(std::fma(1, 1., 1.f) == 2);
    assert(std::fma(1., 1., 1.f) == 2);
    assert(std::fma(1.f, 1., 1.f) == 2);
    assert(std::fma(1, 1.f, 1.f) == 2);
    assert(std::fma(1., 1.f, 1.f) == 2);
    assert(std::fma(1.f, 1.f, 1.f) == 2);
}

__device__ void test_fmax()
{
    static_assert((std::is_same<decltype(std::fmax((float)0, (float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::fmax((bool)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fmax((unsigned short)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fmax((float)0, (unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fmax((double)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fmax((int)0, (long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fmax((int)0, (unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fmax((double)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fmax((float)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fmaxf(0,0)), float>::value), "");
    static_assert((std::is_same<decltype(std::fmax((int)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(fmax(Ambiguous(), Ambiguous())), Ambiguous>::value), "");

    assert(std::fmax(1, 0) == 1);
    assert(std::fmax(1., 0) == 1);
    assert(std::fmax(1.f, 0) == 1);

    assert(std::fmax(1, 0.) == 1);
    assert(std::fmax(1., 0.) == 1);
    assert(std::fmax(1.f, 0.) == 1);

    assert(std::fmax(1, 0.f) == 1);
    assert(std::fmax(1., 0.f) == 1);
    assert(std::fmax(1.f, 0.f) == 1);
}

__device__ void test_fmin()
{
    static_assert((std::is_same<decltype(std::fmin((float)0, (float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::fmin((bool)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fmin((unsigned short)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fmin((float)0, (unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fmin((double)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fmin((int)0, (long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fmin((int)0, (unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fmin((double)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fmin((float)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::fminf(0,0)), float>::value), "");
    static_assert((std::is_same<decltype(std::fmin((int)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(fmin(Ambiguous(), Ambiguous())), Ambiguous>::value), "");

    assert(std::fmin(1, 0) == 0);
    assert(std::fmin(1., 0) == 0);
    assert(std::fmin(1.f, 0) == 0);

    assert(std::fmin(1, 0.) == 0);
    assert(std::fmin(1., 0.) == 0);
    assert(std::fmin(1.f, 0.) == 0);

    assert(std::fmin(1, 0.f) == 0);
    assert(std::fmin(1., 0.f) == 0);
    assert(std::fmin(1.f, 0.f) == 0);
}

__device__ void test_hypot()
{
    static_assert((std::is_same<decltype(std::hypot((float)0, (float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::hypot((bool)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::hypot((unsigned short)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::hypot((float)0, (unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::hypot((double)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::hypot((int)0, (long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::hypot((int)0, (unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::hypot((double)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::hypot((float)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::hypotf(0,0)), float>::value), "");
    static_assert((std::is_same<decltype(std::hypot((int)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(hypot(Ambiguous(), Ambiguous())), Ambiguous>::value), "");

    assert(std::hypot(3, 4) == 5);
    assert(std::hypot(3, 4.) == 5);
    assert(std::hypot(3, 4.f) == 5);

    assert(std::hypot(3., 4) == 5);
    assert(std::hypot(3., 4.) == 5);
    assert(std::hypot(3., 4.f) == 5);

    assert(std::hypot(3.f, 4) == 5);
    assert(std::hypot(3.f, 4.) == 5);
    assert(std::hypot(3.f, 4.f) == 5);

#if TEST_STD_VER > 14
    static_assert((std::is_same<decltype(std::hypot((float)0, (float)0, (float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::hypot((float)0, (bool)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::hypot((float)0, (unsigned short)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::hypot((float)0, (unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::hypot((float)0, (double)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::hypot((float)0, (int)0, (long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::hypot((float)0, (int)0, (unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::hypot((float)0, (double)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::hypot((float)0, (float)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::hypot((int)0, (int)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(hypot(Ambiguous(), Ambiguous(), Ambiguous())), Ambiguous>::value), "");

    assert(std::hypot(2,3,6) == 7);
    assert(std::hypot(1,4,8) == 9);
#endif
}

__device__ void test_ilogb()
{
    static_assert((std::is_same<decltype(std::ilogb((float)0)), int>::value), "");
    static_assert((std::is_same<decltype(std::ilogb((bool)0)), int>::value), "");
    static_assert((std::is_same<decltype(std::ilogb((unsigned short)0)), int>::value), "");
    static_assert((std::is_same<decltype(std::ilogb((int)0)), int>::value), "");
    static_assert((std::is_same<decltype(std::ilogb((unsigned int)0)), int>::value), "");
    static_assert((std::is_same<decltype(std::ilogb((long)0)), int>::value), "");
    static_assert((std::is_same<decltype(std::ilogb((unsigned long)0)), int>::value), "");
    static_assert((std::is_same<decltype(std::ilogb((long long)0)), int>::value), "");
    static_assert((std::is_same<decltype(std::ilogb((unsigned long long)0)), int>::value), "");
    static_assert((std::is_same<decltype(std::ilogb((double)0)), int>::value), "");
    static_assert((std::is_same<decltype(std::ilogbf(0)), int>::value), "");
    // No CUDA ilogbl (takes a long double).
    //static_assert((std::is_same<decltype(std::ilogbl(0)), int>::value), "");
    static_assert((std::is_same<decltype(ilogb(Ambiguous())), Ambiguous>::value), "");
    assert(std::ilogb(1) == 0);
    assert(std::ilogb(1.) == 0);
    assert(std::ilogb(1.f) == 0);
}

__device__ void test_lgamma()
{
    static_assert((std::is_same<decltype(std::lgamma((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::lgamma((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::lgamma((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::lgamma((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::lgamma((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::lgamma((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::lgamma((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::lgamma((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::lgamma((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::lgamma((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::lgammaf(0)), float>::value), "");
    static_assert((std::is_same<decltype(lgamma(Ambiguous())), Ambiguous>::value), "");
    assert(std::lgamma(1) == 0);
}

__device__ void test_llrint()
{
    static_assert((std::is_same<decltype(std::llrint((float)0)), long long>::value), "");
    static_assert((std::is_same<decltype(std::llrint((bool)0)), long long>::value), "");
    static_assert((std::is_same<decltype(std::llrint((unsigned short)0)), long long>::value), "");
    static_assert((std::is_same<decltype(std::llrint((int)0)), long long>::value), "");
    static_assert((std::is_same<decltype(std::llrint((unsigned int)0)), long long>::value), "");
    static_assert((std::is_same<decltype(std::llrint((long)0)), long long>::value), "");
    static_assert((std::is_same<decltype(std::llrint((unsigned long)0)), long long>::value), "");
    static_assert((std::is_same<decltype(std::llrint((long long)0)), long long>::value), "");
    static_assert((std::is_same<decltype(std::llrint((unsigned long long)0)), long long>::value), "");
    static_assert((std::is_same<decltype(std::llrint((double)0)), long long>::value), "");
    static_assert((std::is_same<decltype(std::llrintf(0)), long long>::value), "");
    // No CUDA llrintl (takes a long double).
    //static_assert((std::is_same<decltype(std::llrintl(0)), long long>::value), "");
    static_assert((std::is_same<decltype(llrint(Ambiguous())), Ambiguous>::value), "");
    assert(std::llrint(1) == 1LL);
    assert(std::llrint(1.) == 1LL);
#if CUDA_VERSION > 7050
    assert(std::llrint(1.f) == 1LL);
#endif
}

__device__ void test_llround()
{
    static_assert((std::is_same<decltype(std::llround((float)0)), long long>::value), "");
    static_assert((std::is_same<decltype(std::llround((bool)0)), long long>::value), "");
    static_assert((std::is_same<decltype(std::llround((unsigned short)0)), long long>::value), "");
    static_assert((std::is_same<decltype(std::llround((int)0)), long long>::value), "");
    static_assert((std::is_same<decltype(std::llround((unsigned int)0)), long long>::value), "");
    static_assert((std::is_same<decltype(std::llround((long)0)), long long>::value), "");
    static_assert((std::is_same<decltype(std::llround((unsigned long)0)), long long>::value), "");
    static_assert((std::is_same<decltype(std::llround((long long)0)), long long>::value), "");
    static_assert((std::is_same<decltype(std::llround((unsigned long long)0)), long long>::value), "");
    static_assert((std::is_same<decltype(std::llround((double)0)), long long>::value), "");
    static_assert((std::is_same<decltype(std::llroundf(0)), long long>::value), "");
    // No CUDA llroundl.
    //static_assert((std::is_same<decltype(std::llroundl(0)), long long>::value), "");
    static_assert((std::is_same<decltype(llround(Ambiguous())), Ambiguous>::value), "");
    assert(std::llround(1) == 1LL);
    assert(std::llround(1.) == 1LL);
    assert(std::llround(1.f) == 1LL);
}

__device__ void test_log1p()
{
    static_assert((std::is_same<decltype(std::log1p((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::log1p((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log1p((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log1p((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log1p((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log1p((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log1p((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log1p((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log1p((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log1p((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log1pf(0)), float>::value), "");
    static_assert((std::is_same<decltype(log1p(Ambiguous())), Ambiguous>::value), "");
    assert(std::log1p(0) == 0);
    assert(std::log1p(0.) == 0);
    assert(std::log1p(0.f) == 0);
}

__device__ void test_log2()
{
    static_assert((std::is_same<decltype(std::log2((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::log2((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log2((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log2((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log2((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log2((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log2((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log2((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log2((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log2((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::log2f(0)), float>::value), "");
    static_assert((std::is_same<decltype(log2(Ambiguous())), Ambiguous>::value), "");
    assert(std::log2(1) == 0);
    assert(std::log2(1.) == 0);
    assert(std::log2(1.f) == 0);
}

__device__ void test_logb()
{
    static_assert((std::is_same<decltype(std::logb((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::logb((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::logb((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::logb((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::logb((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::logb((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::logb((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::logb((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::logb((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::logb((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::logbf(0)), float>::value), "");
    static_assert((std::is_same<decltype(logb(Ambiguous())), Ambiguous>::value), "");
    assert(std::logb(1) == 0);
    assert(std::logb(1.) == 0);
    assert(std::logb(1.f) == 0);
}

__device__ void test_lrint()
{
    static_assert((std::is_same<decltype(std::lrint((float)0)), long>::value), "");
    static_assert((std::is_same<decltype(std::lrint((bool)0)), long>::value), "");
    static_assert((std::is_same<decltype(std::lrint((unsigned short)0)), long>::value), "");
    static_assert((std::is_same<decltype(std::lrint((int)0)), long>::value), "");
    static_assert((std::is_same<decltype(std::lrint((unsigned int)0)), long>::value), "");
    static_assert((std::is_same<decltype(std::lrint((long)0)), long>::value), "");
    static_assert((std::is_same<decltype(std::lrint((unsigned long)0)), long>::value), "");
    static_assert((std::is_same<decltype(std::lrint((long long)0)), long>::value), "");
    static_assert((std::is_same<decltype(std::lrint((unsigned long long)0)), long>::value), "");
    static_assert((std::is_same<decltype(std::lrint((double)0)), long>::value), "");
    static_assert((std::is_same<decltype(std::lrintf(0)), long>::value), "");
    // No CUDA lrintl (takes a long double).
    //static_assert((std::is_same<decltype(std::lrintl(0)), long>::value), "");
    static_assert((std::is_same<decltype(lrint(Ambiguous())), Ambiguous>::value), "");
    assert(std::lrint(1) == 1L);
    assert(std::lrint(1.) == 1L);
#if CUDA_VERSION > 7050
    assert(std::lrint(1.f) == 1L);
#endif
}

__device__ void test_lround()
{
    static_assert((std::is_same<decltype(std::lround((float)0)), long>::value), "");
    static_assert((std::is_same<decltype(std::lround((bool)0)), long>::value), "");
    static_assert((std::is_same<decltype(std::lround((unsigned short)0)), long>::value), "");
    static_assert((std::is_same<decltype(std::lround((int)0)), long>::value), "");
    static_assert((std::is_same<decltype(std::lround((unsigned int)0)), long>::value), "");
    static_assert((std::is_same<decltype(std::lround((long)0)), long>::value), "");
    static_assert((std::is_same<decltype(std::lround((unsigned long)0)), long>::value), "");
    static_assert((std::is_same<decltype(std::lround((long long)0)), long>::value), "");
    static_assert((std::is_same<decltype(std::lround((unsigned long long)0)), long>::value), "");
    static_assert((std::is_same<decltype(std::lround((double)0)), long>::value), "");
    static_assert((std::is_same<decltype(std::lroundf(0)), long>::value), "");
    // No CUDA lroundl (takes a long double).
    //static_assert((std::is_same<decltype(std::lroundl(0)), long>::value), "");
    static_assert((std::is_same<decltype(lround(Ambiguous())), Ambiguous>::value), "");
    assert(std::lround(1) == 1L);
    assert(std::lround(1.) == 1L);
    assert(std::lround(1.f) == 1L);
}

__device__ void test_nan()
{
    static_assert((std::is_same<decltype(std::nan("")), double>::value), "");
    static_assert((std::is_same<decltype(std::nanf("")), float>::value), "");
}

__device__ void test_nearbyint()
{
    static_assert((std::is_same<decltype(std::nearbyint((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::nearbyint((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::nearbyint((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::nearbyint((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::nearbyint((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::nearbyint((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::nearbyint((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::nearbyint((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::nearbyint((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::nearbyint((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::nearbyintf(0)), float>::value), "");
    static_assert((std::is_same<decltype(nearbyint(Ambiguous())), Ambiguous>::value), "");
    assert(std::nearbyint(1) == 1);
    assert(std::nearbyint(1.) == 1);
    assert(std::nearbyint(1.f) == 1);
}

__device__ void test_nextafter()
{
    static_assert((std::is_same<decltype(std::nextafter((float)0, (float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::nextafter((bool)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::nextafter((unsigned short)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::nextafter((float)0, (unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::nextafter((double)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::nextafter((int)0, (long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::nextafter((int)0, (unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::nextafter((double)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::nextafter((float)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::nextafterf(0,0)), float>::value), "");
    static_assert((std::is_same<decltype(std::nextafter((int)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(nextafter(Ambiguous(), Ambiguous())), Ambiguous>::value), "");

    // Invoke all our overloads, even if we can't be bothered to check the
    // results.
    std::nextafter(0, 1);
    std::nextafter(0, 1.);
    std::nextafter(0, 1.f);

    std::nextafter(0., 1);
    std::nextafter(0., 1.);
    std::nextafter(0., 1.f);

    std::nextafter(0.f, 1);
    std::nextafter(0.f, 1.);
    std::nextafter(0.f, 1.f);
}

__device__ void test_nexttoward()
{
    static_assert((std::is_same<decltype(nexttoward(Ambiguous(), Ambiguous())), Ambiguous>::value), "");

    // Invoke all our overloads, even if we can't be bothered to check the
    // results.
    std::nexttoward(0, 1);
    std::nexttoward(0, 1.);
    std::nexttoward(0, 1.f);

    std::nexttoward(0., 1);
    std::nexttoward(0., 1.);
    std::nexttoward(0., 1.f);

    std::nexttoward(0.f, 1);
    std::nexttoward(0.f, 1.);
    std::nexttoward(0.f, 1.f);
}

__device__ void test_remainder()
{
    static_assert((std::is_same<decltype(std::remainder((float)0, (float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::remainder((bool)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::remainder((unsigned short)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::remainder((float)0, (unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::remainder((double)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::remainder((int)0, (long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::remainder((int)0, (unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::remainder((double)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::remainder((float)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::remainderf(0,0)), float>::value), "");
    static_assert((std::is_same<decltype(std::remainder((int)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(remainder(Ambiguous(), Ambiguous())), Ambiguous>::value), "");

    assert(std::remainder(1.5, 1) == -.5);
    assert(std::remainder(1.5, 1.) == -.5);
    assert(std::remainder(1.5, 1.f) == -.5);

    assert(std::remainder(1.5f, 1) == -.5);
    assert(std::remainder(1.5f, 1.) == -.5);
    assert(std::remainder(1.5f, 1.f) == -.5);

    assert(std::remainder(2, 1) == 0);
    assert(std::remainder(2, 1.) == 0);
    assert(std::remainder(2, 1.f) == 0);
}

__device__ void test_remquo()
{
    int ip;
    static_assert((std::is_same<decltype(std::remquo((float)0, (float)0, &ip)), float>::value), "");
    static_assert((std::is_same<decltype(std::remquo((bool)0, (float)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(std::remquo((unsigned short)0, (double)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(std::remquo((float)0, (unsigned int)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(std::remquo((double)0, (long)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(std::remquo((int)0, (long long)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(std::remquo((int)0, (unsigned long long)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(std::remquo((double)0, (double)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(std::remquo((float)0, (double)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(std::remquof(0,0, &ip)), float>::value), "");
    static_assert((std::is_same<decltype(std::remquo((int)0, (int)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(remquo(Ambiguous(), Ambiguous(), &ip)), Ambiguous>::value), "");

    assert(std::remquo(1, 1, &ip) == 0);
    assert(std::remquo(1, 1., &ip) == 0);
    assert(std::remquo(1, 1.f, &ip) == 0);

    assert(std::remquo(0.5, 1, &ip) == 0.5);
    assert(std::remquo(0.5, 1., &ip) == 0.5);
    assert(std::remquo(0.5, 1.f, &ip) == 0.5);

    assert(std::remquo(0.5f, 1, &ip) == 0.5);
    assert(std::remquo(0.5f, 1., &ip) == 0.5);
    assert(std::remquo(0.5f, 1.f, &ip) == 0.5);
}

__device__ void test_rint()
{
    static_assert((std::is_same<decltype(std::rint((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::rint((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::rint((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::rint((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::rint((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::rint((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::rint((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::rint((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::rint((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::rint((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::rintf(0)), float>::value), "");
    static_assert((std::is_same<decltype(rint(Ambiguous())), Ambiguous>::value), "");
    assert(std::rint(1) == 1);
    assert(std::rint(1.) == 1);
    assert(std::rint(1.f) == 1);
}

__device__ void test_round()
{
    static_assert((std::is_same<decltype(std::round((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::round((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::round((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::round((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::round((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::round((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::round((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::round((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::round((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::round((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::roundf(0)), float>::value), "");
    static_assert((std::is_same<decltype(round(Ambiguous())), Ambiguous>::value), "");
    assert(std::round(1) == 1);
    assert(std::round(1.) == 1);
    assert(std::round(1.f) == 1);
}

__device__ void test_scalbln()
{
    static_assert((std::is_same<decltype(std::scalbln((float)0, (long)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::scalbln((bool)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::scalbln((unsigned short)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::scalbln((int)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::scalbln((unsigned int)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::scalbln((long)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::scalbln((unsigned long)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::scalbln((long long)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::scalbln((unsigned long long)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::scalbln((double)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::scalblnf(0, (long)0)), float>::value), "");
    static_assert((std::is_same<decltype(scalbln(Ambiguous(), Ambiguous())), Ambiguous>::value), "");
    assert(std::scalbln(1, 1) == 2);
    assert(std::scalbln(1, 1.) == 2);
    assert(std::scalbln(1, 1.f) == 2);

    assert(std::scalbln(1., 1) == 2);
    assert(std::scalbln(1., 1.) == 2);
    assert(std::scalbln(1., 1.f) == 2);

    assert(std::scalbln(1.f, 1) == 2);
    assert(std::scalbln(1.f, 1.) == 2);
    assert(std::scalbln(1.f, 1.f) == 2);
}

__device__ void test_scalbn()
{
    static_assert((std::is_same<decltype(std::scalbn((float)0, (int)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::scalbn((bool)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::scalbn((unsigned short)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::scalbn((int)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::scalbn((unsigned int)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::scalbn((long)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::scalbn((unsigned long)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::scalbn((long long)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::scalbn((unsigned long long)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::scalbn((double)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::scalbnf(0, (int)0)), float>::value), "");
    static_assert((std::is_same<decltype(scalbn(Ambiguous(), Ambiguous())), Ambiguous>::value), "");
    assert(std::scalbn(1, 1) == 2);
    assert(std::scalbn(1, 1.) == 2);
    assert(std::scalbn(1, 1.f) == 2);

    assert(std::scalbn(1., 1) == 2);
    assert(std::scalbn(1., 1.) == 2);
    assert(std::scalbn(1., 1.f) == 2);

    assert(std::scalbn(1.f, 1) == 2);
    assert(std::scalbn(1.f, 1.) == 2);
    assert(std::scalbn(1.f, 1.f) == 2);
}

__device__ void test_tgamma()
{
    static_assert((std::is_same<decltype(std::tgamma((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::tgamma((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::tgamma((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::tgamma((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::tgamma((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::tgamma((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::tgamma((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::tgamma((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::tgamma((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::tgamma((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::tgammaf(0)), float>::value), "");
    static_assert((std::is_same<decltype(tgamma(Ambiguous())), Ambiguous>::value), "");
    assert(std::tgamma(1) == 1);
    assert(std::tgamma(1.) == 1);
    assert(std::tgamma(1.f) == 1);
}

__device__ void test_trunc()
{
    static_assert((std::is_same<decltype(std::trunc((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(std::trunc((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::trunc((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::trunc((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::trunc((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::trunc((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::trunc((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::trunc((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::trunc((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::trunc((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(std::truncf(0)), float>::value), "");
    static_assert((std::is_same<decltype(trunc(Ambiguous())), Ambiguous>::value), "");
    assert(std::trunc(1) == 1);
    assert(std::trunc(1.) == 1);
    assert(std::trunc(1.f) == 1);
}

__global__ void tests()
{
    test_abs();
    test_acos();
    test_asin();
    test_atan();
    test_atan2();
    test_ceil();
    test_cos();
    test_cosh();
    test_exp();
    test_fabs();
    test_floor();
    test_fmod();
    test_frexp();
    test_ldexp();
    test_log();
    test_log10();
    test_modf();
    test_pow();
    test_sin();
    test_sinh();
    test_sqrt();
    test_tan();
    test_tanh();
    test_signbit();
    test_fpclassify();
    test_isfinite();
    test_isnormal();
    test_isgreater();
    test_isgreaterequal();
    test_isinf();
    test_isless();
    test_islessequal();
    test_islessgreater();
    test_isnan();
    test_isunordered();
    test_acosh();
    test_asinh();
    test_atanh();
    test_cbrt();
    test_copysign();
    test_erf();
    test_erfc();
    test_exp2();
    test_expm1();
    test_fdim();
    test_fma();
    test_fmax();
    test_fmin();
    test_hypot();
    test_ilogb();
    test_lgamma();
    test_llrint();
    test_llround();
    test_log1p();
    test_log2();
    test_logb();
    test_lrint();
    test_lround();
    test_nan();
    test_nearbyint();
    test_nextafter();
    test_nexttoward();
    test_remainder();
    test_remquo();
    test_rint();
    test_round();
    test_scalbln();
    test_scalbn();
    test_tgamma();
    test_trunc();
}

int main() {
    tests<<<1,1>>>();
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
      printf("CUDA error %d\n", (int)err);
      return 1;
    }
    printf("Success!\n");
    return 0;
}

#else

#include <stdio.h>

// No C++11; test is a nop.
int main() {
  printf("Success!\n");
  return 0;
}

#endif // __cplusplus < 201103L
